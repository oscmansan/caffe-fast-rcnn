
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layers/cudnn_softmax_layer.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
void CuDNNSoftmaxLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnSoftmaxForward(handle_, CUDNN_SOFTMAX_ACCURATE,
        CUDNN_SOFTMAX_MODE_CHANNEL,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data));
}

template <typename Dtype, typename Mtype>
void CuDNNSoftmaxLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype,Mtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

    CUDNN_CHECK(cudnnSoftmaxBackward(handle_, CUDNN_SOFTMAX_ACCURATE,
          CUDNN_SOFTMAX_MODE_CHANNEL,
          cudnn::dataType<Dtype>::one,
          top_desc_, top_data, top_desc_, top_diff,
          cudnn::dataType<Dtype>::zero,
          bottom_desc_, bottom_diff));
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNSoftmaxLayer);

}  // namespace caffe
#endif
